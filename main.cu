#include<stdio.h>
#include<iostream>
#include<malloc.h>
#include<ctime>
#include<hip/hip_runtime.h>
#include<assert.h>
using namespace std;
//__constant__ int* device_array;

__global__ void test_clock(int &delay, int &add){
   int threadID = (blockIdx.x * blockDim.x) + threadIdx.x;
   clock_t start=0;
   if(threadID == 0) start = clock();
   for(int k=0;k<100;k++){
      for(int j =0;j<10;j++){
         for(int i=0;i<100;i++){
            if(threadID==0){add+=i;}
	    //add+=j;
	    //if(threadID<11){add+=k;}
		 add+=k;
         }
      }
   }
   if(threadID==0){
       clock_t end = clock();
       delay = (int)(end - start);
   }
}



__global__ void sequence_read(long long int &latency, int* device_array, int n, int access_number){
   extern __shared__ int shared_array[];
   for(int i=0;i<n;i++){shared_array[i]=device_array[i];}
   int* j = &shared_array[0];
   //for(int i=0;i<access_number;i++){j=*(int **)j;}
   //j = &shared_array[0];
   long long int temp = clock64();
   for(int i=0;i<access_number;i++){j=*(int **)j;}
   latency = clock64() - temp;
}
__global__ void static_sequence_read(int &latency, long long unsigned* device_array, int access_number, long long unsigned &last_access_value, int array_size){
   int threadID = (blockIdx.x * blockDim.x) + threadIdx.x;
   long long unsigned *j;
   if(threadID == 0){
       for(int i=0;i<array_size;i++){
          last_access_value = device_array[i];
       }
   }
   j = &device_array[0];
   __syncthreads();//finish intializing the array
   clock_t temp=0;
   if(threadID == 0){temp = clock();}//start clocking
   for(int i=0;i<access_number;i++){if(threadID == 0) j=*(long long unsigned **)j;}//access the data array
   if(threadID == 0){
	   latency = (int)(clock() - temp);
	   last_access_value = j[0];
   }
}
__global__ void static_sequence_read_multism(int* latency, long long unsigned* device_array, int access_number, long long unsigned* last_access_value, int array_size){
   //int threadID = (blockIdx.x * blockDim.x) + threadIdx.x;
   int threadx =threadIdx.x;
   int smid = blockIdx.x;
   clock_t start, end;
   long long unsigned temp_value;
   long long unsigned *j;
   if(threadx == 0){
       for(int i=0;i<array_size;i++){
          temp_value = device_array[i+array_size*smid];
       }
   }
   last_access_value[smid]=temp_value;
   j = &(device_array[array_size*smid]);
   __syncthreads();//finish intializing the array
   //start=0;
   if(threadx == 0){start = clock();}//start clocking
   for(int i=0;i<access_number;i++){if(threadx == 0) j=*(long long unsigned **)j;}//access the data array
   if(threadx == 0){
	   end = clock();
	   latency[smid] = (int)(end - start);
	   last_access_value[smid] = j[0];
   }
}

/*
__global__ void static_sequence_read_noinitialize(int* latency, long long unsigned* device_array, int access_number, long long unsigned* last_access_value, int array_size){
   int threadID = (blockIdx.x * blockDim.x) + threadIdx.x;
   long long unsigned *j;
   
   if(threadID == 0){
       for(int i=0;i<array_size;i++){
          last_access_value = device_array[i];
       }
   }
   j = &device_array[0];
   __syncthreads();//finish intializing the array
   clock_t temp=0;
   if(threadID == 0){temp = clock();}//start clocking
   for(int i=0;i<access_number;i++){if(threadID == 0) j=*(long long unsigned **)j;}//access the data array
   if(threadID == 0){
	   latency = (int)(clock() - temp);
	   last_access_value = j[0];
   }
}
*/
int main(void){
   FILE* fp = fopen("./programout.txt","w");
   assert(fp!=NULL);

/*
   for(int array_size = 64; array_size<2048;array_size+=8){
     int device_size = sizeof(int)*array_size;
     int* device_array;
     int* host_array = (int*)malloc(array_size*sizeof(int*));
     cudaMalloc((void**)&device_array,device_size);
     int stride = 4;
     for(int i = 0; i < array_size; i++){
         int t = i + stride;
         if(t >= array_size) t %= stride;
         host_array[i] = *((int*)(&device_array)) + 4*t;//converse the device from int* to int; 4 is the byte size of an int type
     }
     long long int* timing = (long long int*)malloc(sizeof(long long int));
     long long int* timing_d;
     cudaMalloc((void**)&timing_d, sizeof(long long int));
     printf("start computing!\n");
     cudaMemcpy(device_array,host_array,device_size,cudaMemcpyHostToDevice);
     sequence_read<<<1,1,array_size*sizeof(int)>>>(timing_d[0], device_array, array_size, 1000000);
     cudaMemcpy(timing,timing_d,sizeof(long long int),cudaMemcpyDeviceToHost);
     printf ("It took me %lld clicks.\n",timing[0]);
     delete host_array;
     //printf ("It took me %Lf clicks.\n",timing[0]);
   }
*/
	/*
	//cudaEvent_t event1, event2;
	//cudaEventCreate(&event1);
	//cudaEventCreate(&event2);
	
	int* d_time;
	int time;
     int add = 0;
     int* d_add;printf("%d,%d\n",time,add);
     cudaMalloc((void**)&d_time,sizeof(int));
     cudaMalloc((void**)&d_add,sizeof(int));
     cudaMemcpy(d_add,&add,sizeof(int),cudaMemcpyHostToDevice); 
     clock_t start = clock();
     //cudaEventRecord(event1 ,0);
     test_clock<<<1,1>>>(d_time[0],d_add[0]);
     //cudaEventRecord(event2,0);
     //cudaEventSynchronize(event1);
     //cudaEventSynchronize(event2);
     //cudaDeviceSynchronize();
     clock_t end = clock();
     cudaMemcpy(&time,d_time,sizeof(int),cudaMemcpyDeviceToHost);
     cudaMemcpy(&add,d_add,sizeof(int),cudaMemcpyDeviceToHost);
     long double time_elapsed_ms = 1000.0 * (end-start) / CLOCKS_PER_SEC;
     cout << "CPU time used: " << time_elapsed_ms << " ms\n";
     printf("%d,%d\n",time,add);
     //float dt_ms;
     //cudaEventElapsedTime(&dt_ms, event1, event2);
     //cout << "cuda event elpased time:" << dt_ms << " ms\n";
*/
     for(long long unsigned array_size = 16; array_size < 20; array_size += 4){
     int sm_max = 1;
     //long long unsigned array_size = 16;
     //printf("array size =%d\n",array_size);
     fprintf(fp,"%d\t",array_size);
     long long unsigned device_size = sizeof(long long unsigned)*array_size*sm_max;
     long long unsigned* device_array;
     long long unsigned* host_array = (long long unsigned*)malloc(array_size*sizeof(long long unsigned*)*sm_max);
     printf("Strat mallocing\n");
     assert(hipSuccess == hipMalloc((void**)&device_array,device_size));
     int stride = 16;//set the access stride = cache_line_size / sizeof(long long unsigned) = 128/8=16
     for(int sm_id =0;sm_id<sm_max;sm_id++){
         for(int i = 0; i < array_size; i++){
             int t = i + stride;
             if(t >= array_size) t %= stride;
             host_array[i+array_size*sm_id] = (long long unsigned)(&(device_array[sm_id*array_size])) + sizeof(long long unsigned)*t;//converse the device from int* to int; 4 is the byte size of an int type
         }
     }
   

/*
     cout<< "sizeof long long unsigned" << sizeof(long long unsigned) << endl;
     cout<< "device array adress: " << (long long unsigned)device_array << endl;
     for(int i=0;i<array_size;i++){
         cout << host_array[i] << endl;
     }
     return 0;
*/


     int* timing = (int*)malloc(sizeof(int)*sm_max);
     int* timing_d;
    // printf ("It took me %d clicks before the funvtion call.\n",timing[0]);
     assert(hipSuccess == hipMalloc((void**)&timing_d, sizeof(int)*sm_max));
     long long unsigned* last_access_value = (long long unsigned*)malloc(sizeof(long long unsigned)*sm_max);
     long long unsigned* d_last_access_value;
    // printf ("original last_access value: %llu\n", last_access_value[0]);
     assert(hipSuccess == hipMalloc((void**)&d_last_access_value, sizeof(long long unsigned)*sm_max));
    // printf("start computing!\n");
     assert(hipSuccess == hipMemcpy(device_array,host_array,device_size,hipMemcpyHostToDevice));
    
double access_time;
    /* 
     cudaDeviceSynchronize();
     static_sequence_read_multism<<<sm_max,1>>>(timing_d, device_array, 4, d_last_access_value, array_size);
     cudaDeviceSynchronize();
     assert(cudaSuccess == cudaMemcpy(timing,timing_d,sizeof(int)*sm_max,cudaMemcpyDeviceToHost));
     assert(cudaSuccess == cudaMemcpy(last_access_value,d_last_access_value,sizeof(long long unsigned)*sm_max,cudaMemcpyDeviceToHost));
     cudaDeviceSynchronize();
     access_time=0;
     for(int i=0;i<sm_max;i++){
     //printf ("It took me %d clicks, last_access value: %llu.\n",timing[i], last_access_value[i]);
         access_time+=timing[i];
     }
     printf("It took me %lf clicks",access_time/sm_max);
    */



     hipDeviceSynchronize();
     static_sequence_read_multism<<<sm_max,1>>>(timing_d, device_array, 16, d_last_access_value, array_size);
     hipDeviceSynchronize();
     assert(hipSuccess == hipMemcpy(timing,timing_d,sizeof(int)*sm_max,hipMemcpyDeviceToHost));
     assert(hipSuccess == hipMemcpy(last_access_value,d_last_access_value,sizeof(long long unsigned)*sm_max,hipMemcpyDeviceToHost));
     hipDeviceSynchronize();
     access_time = 0;
     for(int i=0;i<sm_max;i++){
     //printf ("It took me %d clicks, last_access value: %llu.\n",timing[i], last_access_value[i]);
         access_time+=timing[i];
     }
    // printf("It took me %lf clicks",access_time/sm_max);
     fprintf(fp,"%lf\n",access_time/sm_max);
     fclose(fp);


     /*
     cudaDeviceSynchronize();
     static_sequence_read_multism<<<sm_max,1>>>(timing_d, device_array, 1, d_last_access_value, array_size);
     cudaDeviceSynchronize();
     assert(cudaSuccess == cudaMemcpy(timing,timing_d,sizeof(int)*sm_max,cudaMemcpyDeviceToHost));
     assert(cudaSuccess == cudaMemcpy(last_access_value,d_last_access_value,sizeof(long long unsigned)*sm_max,cudaMemcpyDeviceToHost));
     cudaDeviceSynchronize();
     access_time = 0;
     for(int i=0;i<sm_max;i++){
     //printf ("It took me %d clicks, last_access value: %llu.\n",timing[i], last_access_value[i]);
         access_time+=timing[i];
     }
     printf("It took me %lf clicks",access_time/sm_max);
     printf("\n");
*/


/*
     static_sequence_read<<<1,1>>>(timing_d[0], device_array, 32, d_last_access_value[0], array_size);
     assert(cudaSuccess == cudaMemcpy(timing,timing_d,sizeof(int),cudaMemcpyDeviceToHost));
     assert(cudaSuccess == cudaMemcpy(last_access_value,d_last_access_value,sizeof(long long unsigned),cudaMemcpyDeviceToHost));
     cudaDeviceSynchronize();
     printf ("It took me %d clicks, last_access value: %llu.\n",timing[0], last_access_value[0]);
     
     static_sequence_read<<<1,1>>>(timing_d[0], device_array, 128, d_last_access_value[0], array_size);
     assert(cudaSuccess == cudaMemcpy(timing,timing_d,sizeof(int),cudaMemcpyDeviceToHost));
     assert(cudaSuccess == cudaMemcpy(last_access_value,d_last_access_value,sizeof(long long unsigned),cudaMemcpyDeviceToHost));
     cudaDeviceSynchronize();
     printf ("It took me %d clicks, last_access value: %llu.\n",timing[0], last_access_value[0], array_size);
*/
    /* static_sequence_read_noinitialize<<<1,1>>>(timing_d[0], device_array, 4096, d_last_access_value[0], array_size);
     assert(cudaSuccess == cudaMemcpy(timing,timing_d,sizeof(int),cudaMemcpyDeviceToHost));
     assert(cudaSuccess == cudaMemcpy(last_access_value,d_last_access_value,sizeof(long long unsigned),cudaMemcpyDeviceToHost));
     cudaDeviceSynchronize();
     printf ("no initiliaze took %d clicks, last_access value: %llu.\n",timing[0], last_access_value[0]);
*/
     /*
     static_sequence_read<<<1,1>>>(timing_d[0], device_array, 4096, d_last_access_value[0], array_size);
     assert(cudaSuccess == cudaMemcpy(timing,timing_d,sizeof(int),cudaMemcpyDeviceToHost));
     assert(cudaSuccess == cudaMemcpy(last_access_value,d_last_access_value,sizeof(long long unsigned),cudaMemcpyDeviceToHost));
     cudaDeviceSynchronize();
     printf ("It took me %d clicks, last_access value: %llu.\n",timing[0], last_access_value[0]);
  */
     /*   
     static_sequence_read_noinitialize<<<1,1>>>(timing_d[0], device_array, 4096, d_last_access_value[0], array_size);
     assert(cudaSuccess == cudaMemcpy(timing,timing_d,sizeof(int),cudaMemcpyDeviceToHost));
     assert(cudaSuccess == cudaMemcpy(last_access_value,d_last_access_value,sizeof(long long unsigned),cudaMemcpyDeviceToHost));
     cudaDeviceSynchronize();
     printf ("no initiliaze took %d clicks, last_access value: %llu.\n",timing[0], last_access_value[0]);
*/
     /*
     static_sequence_read<<<1,1>>>(timing_d[0], device_array, 4096, d_last_access_value[0], array_size);
     assert(cudaSuccess == cudaMemcpy(timing,timing_d,sizeof(int),cudaMemcpyDeviceToHost));
     assert(cudaSuccess == cudaMemcpy(last_access_value,d_last_access_value,sizeof(long long unsigned),cudaMemcpyDeviceToHost));
     cudaDeviceSynchronize();
     printf ("It took me %d clicks, last_access value: %llu.\n",timing[0], last_access_value[0]);
     delete host_array;
     printf ("\n");
     */
     }
   return 0;
} 





